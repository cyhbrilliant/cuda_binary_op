#include <iostream>
#include <stdio.h>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;



void MatrixRandBin(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if ((float)rand()/RAND_MAX > 0.5) {
                mat[i*cols+j] = 1.0f;
            }else {
                mat[i*cols+j] = -1.0f;
            }

        }
    }
}

void MatrixPrint(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << setw(2) << mat[i*cols+j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

void MatrixPrintD(int *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << setw(2) << mat[i*cols+j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}


float MatrixCompare(float *a, float *b, int rows, int cols) {
    float err = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            err += abs(a[i*cols+j]-b[i*cols+j]);  
        }
    }
    return err;
}

void MatrixMul_host(float *a, int a_rows, int a_cols, float *b, int b_rows, int b_cols, float *c) {
    for (int i = 0; i < a_rows; i++) {
        for (int j = 0; j < b_cols; j++) {
            float t = 0;
            for (int k = 0; k < b_rows; k++) {
                t += a[i*a_cols+k]*b[k*b_cols+j];
            }
            c[i*b_cols+j] = t;
        }
    }
}

//horizontal
__global__ void AMatrix2Bin(float *a, int *a_bin, int pitch_a, int Pitch_a_bin, int a_rows, int MaxBlocks, int BINSIZE) {
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    int bdx = blockDim.x;
    int gdx = gridDim.x;


    int maxThreads = MaxBlocks*a_rows;
    for (int id = bix*bdx+tix; id < maxThreads; id += gdx*bdx) {
        int rid = id/MaxBlocks;
        int cid = id%MaxBlocks;

        int Integer = 0;
        int base = 1;
        for (int i = 0; i < BINSIZE; i++) {
            if (a[rid*pitch_a+(cid+1)*BINSIZE-1-i] == 1.f) {
                Integer += base;
            }
            base = base<<1;
        }

        a_bin[rid*Pitch_a_bin+cid] = Integer;
    }

}
//vetical
__global__ void BMatrix2Bin(float *b, int *b_bin, int pitch_b, int Pitch_b_bin, int b_cols, int MaxBlocks, int BINSIZE) {
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    int bdx = blockDim.x;
    int gdx = gridDim.x;

    int maxThreads = MaxBlocks*b_cols;
    for (int id = bix*bdx+tix; id < maxThreads; id += gdx*bdx) {
        int cid = id/MaxBlocks;
        int rid = id%MaxBlocks;

        int Integer = 0;
        int base = 1;
        for (int i=0; i < BINSIZE; i++) {
            if (b[((rid+1)*BINSIZE-1-i)*pitch_b+cid] == 1.f) {
                Integer += base;
            }
            base = base<<1;
        }

        b_bin[rid*Pitch_b_bin+cid] = Integer;
    }

}

// __device__ unsigned char __popcount_tab_copy[256];//__constant__ is slower than __device__
// __device__ int popcount (int x) {
//   return __popcount_tab_copy[(x >>  0) & 0xff]  
//   + __popcount_tab_copy[(x >>  8) & 0xff]  
//   + __popcount_tab_copy[(x >> 16) & 0xff] 
//   + __popcount_tab_copy[(x >> 24) & 0xff];
// }


//x is cols, y is rows!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
__global__ void MatrixMulXnor(int *a, int *b, float *result, unsigned char *__popcount_tab,
 	int pitch_a, int pitch_b, int pitch_result,
	int RectSize_n, int midBlocks, int BINSIZE, int RealMidSize) {

    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int gdx = gridDim.x;
    int gdy = gridDim.y;
    // printf(" block:(%d, %d) thread:(%d, %d)\n",bix,biy,tix,tiy );
    // printf(" gridDim:(%d, %d) blockDim:(%d, %d)\n",gdx,gdy,bdx,bdy );


    int rest = BINSIZE*RectSize_n*midBlocks-RealMidSize;

    __shared__ unsigned char __popcount_tab_shared[256];
    __shared__ int a_rect_shared[8][16];
    __shared__ int b_rect_shared[16][8];

 	for (int i = tiy*bdx+tix; i < 256; i += bdx*bdy) {
         __popcount_tab_shared[i] = __popcount_tab[i];
    }
    __syncthreads();


    int sum = 0;
    for (int i = 0; i < midBlocks; i++) {
    	for (int j = tix; j < RectSize_n; j += bdx) {
    		a_rect_shared[tiy][j] = a[(biy*bdy+tiy)*pitch_a+i*RectSize_n+j];
    	}
    	for (int j = tiy; j < RectSize_n; j += bdy) {
    		b_rect_shared[j][tix] = b[(i*RectSize_n+j)*pitch_b+bix*bdx+tix];
    	}
    	__syncthreads();


    	int bin = 0;
    	bin = a_rect_shared[tiy][0]^b_rect_shared[0][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][1]^b_rect_shared[1][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][2]^b_rect_shared[2][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][3]^b_rect_shared[3][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][4]^b_rect_shared[4][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][5]^b_rect_shared[5][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][6]^b_rect_shared[6][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][7]^b_rect_shared[7][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][8]^b_rect_shared[8][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][9]^b_rect_shared[9][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][10]^b_rect_shared[10][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][11]^b_rect_shared[11][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][12]^b_rect_shared[12][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][13]^b_rect_shared[13][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][14]^b_rect_shared[14][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tiy][15]^b_rect_shared[15][tix];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    __syncthreads();
    }
    result[(biy*bdy+tiy)*pitch_result+bix*bdx+tix] = sum-rest;



    // num=0;
    // int rest=(BINSIZE*a_cols-RealMidSize);
    // for(int i=bix;i<a_rows;i+=gdx){
    //     for(int j=tix;j<b_cols;j+=bdx){
    //         // printf("i=%d ; j=%d\n",i,j);
    //         int sum=0;
    //         for(int k=0;k<a_cols;k++){
    //             int bin=(a_shared[num*a_cols+k]^b[k*pitch_b+j]);
    //             int negnum=popcount(bin);
    //             int posnum=BINSIZE-negnum;
    //             //calculate ignores the rest of BINSIZE if the Matsize can't devided by BINSIZE ,it can cause err
    //             //(10/00)'(01/00) should be 0000 but it is 0011,so 1+1 is trash in the result.and it mislead a_rows*b_cols times. 
    //             sum+=(posnum-negnum);
    //         }
    //         result[i*pitch_result+j]=sum-rest;
    //     }
    //     num++;
    // }


}


void MatrixMul_device(float *a, float *b, int a_rows, int a_cols, int b_cols, float *result) {

    int BINSIZE = 32;//size of bin2int, 32 means 0000 0000 0000 0000 0000 0000 0000 0000
    int MaxBlocks = (a_cols-1)/BINSIZE+1;
    int Copysize = MaxBlocks*BINSIZE;
    
    float *a_copy;//a_rows * Copysize
    float *b_copy;//Copysize * b_cols
    size_t Pitch_a_copy, Pitch_b_copy;
    hipMallocPitch((void**)&a_copy, &Pitch_a_copy, sizeof(float)*Copysize, a_rows);
    hipMallocPitch((void**)&b_copy, &Pitch_b_copy, sizeof(float)*b_cols, Copysize);
    hipMemset(a_copy, 0, Pitch_a_copy*a_rows);
    hipMemset(b_copy, 0, Pitch_b_copy*Copysize);
    hipMemcpy2D(a_copy, Pitch_a_copy, a, sizeof(float)*a_cols, sizeof(float)*a_cols, a_rows, hipMemcpyDeviceToDevice);
    hipMemcpy2D(b_copy, Pitch_b_copy, b, sizeof(float)*b_cols, sizeof(float)*b_cols, a_cols, hipMemcpyDeviceToDevice);

//check oringin
    // float *a_host;
    // float *b_host;
    // a_host = (float*) malloc(sizeof(float) * Copysize * a_rows);
    // b_host = (float*) malloc(sizeof(float) * b_cols * Copysize);
    // cudaMemcpy2D(a_host,sizeof(float) *Copysize, a_copy,Pitch_a_copy,sizeof(float) *Copysize , a_rows,cudaMemcpyDeviceToHost);
    // cudaMemcpy2D(b_host,sizeof(float) *b_cols, b_copy,Pitch_b_copy,sizeof(float) *b_cols , Copysize,cudaMemcpyDeviceToHost);
    // MatrixPrint(a_host,a_rows,Copysize);
    // MatrixPrint(b_host,Copysize,b_cols);

    //rect[8][16]*[16][32]
	int RectSize_x = 8;
	int RectSize_n = 16;
    int RectSize_y = 8;
    dim3 RectBlockNum_a_bin((MaxBlocks-1)/RectSize_n+1, (a_rows-1)/RectSize_y+1, 1);//with block multiply
    dim3 RectBlockNum_b_bin((b_cols-1)/RectSize_x+1, (MaxBlocks-1)/RectSize_n+1, 1);
    int *a_bin;
    int *b_bin;
    size_t Pitch_a_bin, Pitch_b_bin;
    hipMallocPitch((void**)&a_bin , &Pitch_a_bin , sizeof(int)*RectSize_n*RectBlockNum_a_bin.x, RectSize_y*RectBlockNum_a_bin.y);
    hipMallocPitch((void**)&b_bin , &Pitch_b_bin , sizeof(int)*RectSize_x*RectBlockNum_b_bin.x, RectSize_n*RectBlockNum_b_bin.y);
    hipMemset(a_bin, 0, Pitch_a_bin*RectSize_y*RectBlockNum_a_bin.y);
    hipMemset(b_bin, 0, Pitch_b_bin*RectSize_n*RectBlockNum_b_bin.y);
    dim3 BS_BIN(512,1,1);
    dim3 GS_BIN(6,1,1);
    AMatrix2Bin<<< GS_BIN, BS_BIN >>>(a_copy, a_bin, 
        Pitch_a_copy/sizeof(float), Pitch_a_bin/sizeof(int), a_rows, MaxBlocks, BINSIZE);
    BMatrix2Bin<<< GS_BIN, BS_BIN >>>(b_copy, b_bin, 
        Pitch_b_copy/sizeof(float), Pitch_b_bin/sizeof(int), b_cols, MaxBlocks, BINSIZE);
    hipFree(a_copy);
    hipFree(b_copy);
//check bin
    // int *a_host_bin;
    // int *b_host_bin;
    // a_host_bin = (int*) malloc(sizeof(int) *MaxBlocks * a_rows);
    // b_host_bin = (int*) malloc(sizeof(int) *b_cols * MaxBlocks);
    // cudaMemcpy2D(a_host_bin,sizeof(int) *MaxBlocks, a_bin,Pitch_a_bin,sizeof(int) *MaxBlocks , a_rows ,cudaMemcpyDeviceToHost);
    // cudaMemcpy2D(b_host_bin,sizeof(int) *b_cols, b_bin,Pitch_b_bin,sizeof(int) *b_cols , MaxBlocks ,cudaMemcpyDeviceToHost);
    // MatrixPrintD(a_host_bin,a_rows,MaxBlocks);
    // MatrixPrintD(b_host_bin,MaxBlocks,b_cols);


    float *result_bin;//a_rows * b_cols
    size_t Pitch_result_bin;
    hipMallocPitch((void**)&result_bin , &Pitch_result_bin , sizeof(float)*RectSize_x*RectBlockNum_b_bin.x, RectSize_y*RectBlockNum_a_bin.y);

    const unsigned char __popcount_tab[] = {
      0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8,
    };
    unsigned char *__popcount_tab_copy;
    hipMalloc((void**)&__popcount_tab_copy, sizeof(__popcount_tab));
    hipMemcpy(__popcount_tab_copy, __popcount_tab, sizeof(__popcount_tab), hipMemcpyHostToDevice);

    hipEvent_t start_device, stop_device;
    float time_device;
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);
    hipEventRecord(start_device, 0);

    dim3 BS_MM(RectSize_x, RectSize_y, 1);
    dim3 GS_MM(RectBlockNum_b_bin.x, RectBlockNum_a_bin.y, 1);
    MatrixMulXnor<<< GS_MM, BS_MM >>>(a_bin, b_bin, result_bin, __popcount_tab_copy,
     	Pitch_a_bin/sizeof(int), Pitch_b_bin/sizeof(int), Pitch_result_bin/sizeof(float),
     	RectSize_n, RectBlockNum_a_bin.x, BINSIZE, a_cols);

    hipEventRecord( stop_device, 0 );
    hipEventSynchronize( stop_device );
    hipEventElapsedTime( &time_device, start_device, stop_device );
    hipEventDestroy( start_device );
    hipEventDestroy( stop_device );
    cout<<"gputime="<<time_device<<"ms"<<endl;

    hipMemcpy2D(result,sizeof(float) *b_cols, result_bin,Pitch_result_bin,sizeof(float) *b_cols , a_rows ,hipMemcpyDeviceToDevice);

    hipFree(a_bin);
    hipFree(b_bin);
    hipFree(result_bin);
}

int main(){

//simulate pytorch param
    int x = 2000;
    int n = 2000;
    int y = 2000;
    float *a_host;
    float *b_host;
    float *result_host;
    a_host = (float*) malloc(sizeof(float) * x * n);
    b_host = (float*) malloc(sizeof(float) * n * y);
    result_host = (float*) malloc(sizeof(float) * x * y);
    srand(0);
    MatrixRandBin(a_host,x,n);
    MatrixRandBin(b_host,n,y);
    // cout<<MatrixCopysize<<endl;

    float *a_copy;
    float *b_copy;
    float *result_device;
    hipMalloc((void**)&a_copy,sizeof(float) *x * n);
    hipMalloc((void**)&b_copy,sizeof(float) *n * y);
    hipMalloc((void**)&result_device,sizeof(float) *x * y);
    hipMemcpy(a_copy,a_host,sizeof(float) *x * n,hipMemcpyHostToDevice);
    hipMemcpy(b_copy,b_host,sizeof(float) *n * y,hipMemcpyHostToDevice);


    // MatrixPrint(a_host,x,n);
    // MatrixPrint(b_host,n,y);

//run in gpu warp in C code
    MatrixMul_device(a_copy,b_copy,x,n,y,result_device);

    hipMemcpy(result_host, result_device,sizeof(float) *x * y,hipMemcpyDeviceToHost);
    hipFree(a_copy);
    hipFree(b_copy);
    hipFree(result_device);
    // MatrixPrint(result_host,x,y);

// //run in cpu
//     float *result_cpu;
//     result_cpu = (float*) malloc(sizeof(float) * x * y);
//     clock_t start_host = clock();
//     MatrixMul_host(a_host,x,n,b_host,n,y,result_cpu);
//     cout<<"cputime="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
//     // MatrixPrint(result_cpu,x,y);


// //compare value of gpu and cpu
//     float err=MatrixCompare(result_cpu,result_host,x,y);
//     cout<<"err in gpu and cpu = "<<err<<endl;

    return 0;
}